#include "hip/hip_runtime.h"
//"C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v10.2\bin\nvcc.exe"  -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2017\Community\VC\Tools\MSVC\14.16.27023\bin\Hostx86\x64" -o pack main.cu -O3 -m=64 -arch=compute_61 -code=sm_61 -Xptxas -allow-expensive-optimizations=true -Xptxas -v
#include <iostream>
#include <chrono>
#include <fstream>
#include <algorithm>
#include <inttypes.h>
#include <bitset>
#include <iostream>
#include <vector>
#include <map>
#include <iomanip>
#include <fstream>
#include "lcg.h"


DEVICEABLE static int32_t random_next(lcg::Random *random, int bits) {
	*random = (*random * lcg::MULTIPLIER + lcg::ADDEND) & lcg::MASK;
	return (int32_t) (*random >> (48u - bits));
}

DEVICEABLE static int32_t random_next_int(lcg::Random *random, const uint16_t bound) {
	int32_t r = random_next(random, 31);
	const uint16_t m = bound - 1u;
	if ((bound & m) == 0) {
		r = (int32_t) ((bound * (uint64_t) r) >> 31u);
	} else {
		for (int32_t u = r;
			 u - (r = u % bound) + m < 0;
			 u = random_next(random, 31));
	}
	return r;
}

DEVICEABLE static int32_t random_next_int_nonpow(lcg::Random *random, const uint16_t bound) {
	int32_t r = random_next(random, 31);
	const uint16_t m = bound - 1u;
	for (int32_t u = r;
		 u - (r = u % bound) + m < 0;
		 u = random_next(random, 31));
  return r;
}

#define MAX_TREE_SEARCH 12
#define TREES_IN_CHUNK 2

//CHECK
#define WATERFALL_X 116 
#define WATERFALL_Y 76
#define WATERFALL_Z -31


#define POP_CHUNK_X ((WATERFALL_X-8)>>4)
#define POP_CHUNK_Z ((WATERFALL_Z-8)>>4)
#define WATERFALL_X_IN_POPULATION ((WATERFALL_X - 8)&15)
#define WATERFALL_Z_IN_POPULATION ((WATERFALL_Z - 8)&15)



#define TREE1_X (WATERFALL_X_IN_POPULATION - 5)
#define TREE1_Z (WATERFALL_Z_IN_POPULATION - 8)
#define TREE1_HEIGHT 5

#define TREE2_X (WATERFALL_X_IN_POPULATION - 3)
#define TREE2_Z (WATERFALL_Z_IN_POPULATION + 3)
#define TREE2_HEIGHT 5





//Should return a unique tree mask from 8 bit uint
DEVICEABLE static inline uint8_t GetTreeIndex(int innerX, int innerZ) {
	return ( (uint8_t)(innerX == TREE1_X && innerZ == TREE1_Z) << 0) |
			((uint8_t)(innerX == TREE2_X && innerZ == TREE2_Z) << 1) ;
}	



DEVICEABLE static inline bool TreeAtPosMatchesHeight(int innerX, int innerZ, int height) {
	return  ( (innerX == TREE1_X && innerZ == TREE1_Z && height == TREE1_HEIGHT)) ||
			((innerX == TREE2_X && innerZ == TREE2_Z && height == TREE2_HEIGHT));
}









DEVICEABLE static inline bool WaterfallMatch(lcg::Random rand) {
	// yellow flowers
	lcg::advance<774>(rand);
	// red flowers
	if (random_next(&rand, 1) == 0)
		lcg::advance<387>(rand);
	
	// brown mushroom
	if (random_next(&rand, 2) == 0)
		lcg::advance<387>(rand);
	
	// red mushroom
	if (random_next(&rand, 3) == 0)
		lcg::advance<387>(rand);
		
	// reeds
	lcg::advance<830>(rand);
	
	// pumpkins
	if (random_next(&rand, 5) == 0)
		lcg::advance<387>(rand);
	
	
	for (int i = 0; i < 50; i++) {
		bool waterfall_matches = random_next(&rand, 4) == WATERFALL_X_IN_POPULATION;
		waterfall_matches &= random_next_int(&rand, random_next_int_nonpow(&rand, 120) + 8) == WATERFALL_Y;
		waterfall_matches &= random_next(&rand, 4) == WATERFALL_Z_IN_POPULATION;
		if(waterfall_matches)
			return true;
	}
	return false;
}



DEVICEABLE static inline bool CheckChunkSeed(lcg::Random rand) {
	//Include dungeon skip which is 40
	lcg::advance<40+30+3686+3>(rand);
		
	if (random_next_int_nonpow(&rand,10) == 0)
		return false;
	
	uint8_t treeMask = 0;
	int treeCount = 0;
	for (int attempt = 0; attempt < MAX_TREE_SEARCH; attempt++) {
	  int x = lcg::next_int<16>(rand);
	  int z = lcg::next_int<16>(rand);
	  int treeHeight = 4 + lcg::next_int<3>(rand);
	  
	  uint8_t thisMask = GetTreeIndex(x, z);
	  if ((treeCount != TREES_IN_CHUNK) && (thisMask!=0) && ((thisMask&treeMask) == 0)) {
		if (!TreeAtPosMatchesHeight(x, z, treeHeight))
			//continue;
			return false;
		treeMask |= thisMask;
		// successful tree attempt
		treeCount++;
		lcg::advance<16>(rand); // not sure on this number // pretty sure it is
	  } else {
		// failed tree attempt
	  }
	  
	  if (treeCount == TREES_IN_CHUNK) {
		lcg::Random new_rand = rand;
		// test waterfall loop
		if (WaterfallMatch(new_rand))
			return true;
	  }
	}
	return false;
}


DEVICEABLE static inline bool doCheck(uint64_t seed) {
	lcg::Random chunkSeed = (seed ^ lcg::MULTIPLIER) & lcg::MASK;

	int64_t seedA = (((int64_t)lcg::next_long(chunkSeed))/2L)*2L+1L;
	int64_t seedB = (((int64_t)lcg::next_long(chunkSeed))/2L)*2L+1L;
	chunkSeed = ((((int64_t)POP_CHUNK_X) * seedA + ((int64_t)POP_CHUNK_Z) * seedB ^ seed) ^ lcg::MULTIPLIER)&lcg::MASK;
	

	
	//if(CheckChunkSeed(chunkSeed))
	//	return true;
	
	//Simulate 1 clay being spawned
	//lcg::advance<99-30>(chunkSeed);
	if(CheckChunkSeed(chunkSeed))
		return true;
	return false;
}





#define SEEDSPACE ((1LLU<<48)/8+1)




#define BLOCK_SIZE (256)
#define WORK_SIZE_BITS 24
#define SEEDS_PER_CALL ((1ULL << (WORK_SIZE_BITS)) * (BLOCK_SIZE))



__global__ __launch_bounds__(BLOCK_SIZE,2) void doo_bee_do_be_doo_ba(uint64_t offset, uint32_t* count, uint64_t* buffer) {
	uint64_t seed = blockIdx.x * blockDim.x + threadIdx.x + offset;
	if (seed > SEEDSPACE)
		return;
	
	seed *= 8;//Seed is now the time in nanoseconds
	
	seed += 8682522807148012LLU + 16LLU + 1LLU;//MUST UNCOMMENT + 1LLU//Uniquifier has been added to the seed 
	
	seed = (seed ^ lcg::MULTIPLIER) & lcg::MASK;//Make the new random object
	seed = lcg::next_long(seed);//Get the world seed
	
	if (doCheck(seed))
		buffer[atomicAdd(count, 1)] = seed;
	return;
}










#if defined(WIN32) || defined(_WIN32) || defined(__WIN32) && !defined(__CYGWIN__)
	#include <windows.h>
	uint64_t getCurrentTimeMillis() {
		SYSTEMTIME time;
		GetSystemTime(&time);
		return (uint64_t)((time.wSecond * 1000) + time.wMilliseconds);
	}
#else
	#include <sys/time.h>
	uint64_t getCurrentTimeMillis() {
		struct timeval te; 
		gettimeofday(&te, NULL); // get current time
		uint64_t milliseconds = te.tv_sec*1000LL + te.tv_usec/1000; // calculate milliseconds
		return milliseconds;
	}
#endif
#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
	exit(code);
  }
}



uint32_t* count;
uint64_t* buffer;
std::ofstream seeds_output;
uint64_t start_time;

void doStep(uint64_t offset) {
	uint64_t start = getCurrentTimeMillis();
	*count = 0;
	doo_bee_do_be_doo_ba<<< 1ULL << WORK_SIZE_BITS, BLOCK_SIZE>>>(offset, count, buffer);
	GPU_ASSERT(hipPeekAtLastError());
	GPU_ASSERT(hipDeviceSynchronize());
	
	for(uint64_t index = 0; index < *count; index++)
		seeds_output << (buffer[index]& lcg::MASK) << std::endl;
	
	uint64_t end = getCurrentTimeMillis();
	//Not dividing by 1000000 cause dividing by milliseconds is equivalent to dividing by 1000
	std::cout << std::fixed << std::setprecision(2) << "Speed: " << (((double)SEEDS_PER_CALL/(end - start))/1000) << " mill seed/s," << //million seeds per second
		" step took " << (end - start) << " milliseconds," <<
		" seed count: " << *count << "," <<
		" ETA: " << (int)((((double)(SEEDSPACE-offset))/SEEDS_PER_CALL)*(end - start)/1000) << " seconds," <<
		" done " << std::fixed << std::setprecision(2) << (((double)offset/SEEDSPACE)*100) << "%" <<
		std::endl;
	
	
	//exit(0);
	/*
	uint64_t count = 0;
	uint64_t start = getCurrentTimeMillis();
	for(uint64_t seed =0;seed<100000000;seed++) {
		count += doCheck(seed);
	}*/
}

void setup() {
	seeds_output.open("seeds.txt");
	
	hipSetDevice(0);
	GPU_ASSERT(hipPeekAtLastError());
	GPU_ASSERT(hipDeviceSynchronize());
	
	
	GPU_ASSERT(hipMallocManaged(&count, sizeof(*count)));
	GPU_ASSERT(hipPeekAtLastError());
	
	GPU_ASSERT(hipMallocManaged(&buffer, sizeof(*buffer) * (SEEDS_PER_CALL>>5)));
	GPU_ASSERT(hipPeekAtLastError());
}

void done() {
	seeds_output.close();
}

int main() {
	std::cout << "Waterfall pop chunk x: " << POP_CHUNK_X << " waterfall x in pop chunk pos: " << WATERFALL_X_IN_POPULATION << std::endl;
	std::cout << "Waterfall pop chunk z: " << POP_CHUNK_Z << " waterfall z in pop chunk pos: " << WATERFALL_Z_IN_POPULATION << std::endl;
	std::cout << "Seeds per call/Step size: " << SEEDS_PER_CALL << std::endl;
	std::cout << "Seedspace size: " << SEEDSPACE << std::endl;
	std::cout << "Estimated steps: " << (SEEDSPACE/SEEDS_PER_CALL+1) << std::endl;
	
	start_time = getCurrentTimeMillis();
	
	setup();
	std::cout << "Starting now" << std::endl;
	for(uint64_t offset = 0; offset < (SEEDSPACE + SEEDS_PER_CALL); offset += SEEDS_PER_CALL)
		doStep(offset);
	done();
	hipFree(count);
	GPU_ASSERT(hipPeekAtLastError());
	hipFree(buffer);
	GPU_ASSERT(hipPeekAtLastError());
	
	std::cout << "Finished in " << ((getCurrentTimeMillis() - start_time)/1000) << " seconds" <<std::endl;
	return 0;
}







