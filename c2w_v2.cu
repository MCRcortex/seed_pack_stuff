#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include <algorithm>
#include <inttypes.h>
#include <bitset>
#include <iostream>
#include <vector>
#include <map>
#include <iomanip>
#include <fstream>
#include <chrono>
#include <mutex>
uint64_t millis() {return (std::chrono::duration_cast< std::chrono::milliseconds >(std::chrono::system_clock::now().time_since_epoch())).count();}



#define INPUT_FILE "seeds.txt"
#define OUTPUT_FILE "world_seeds.txt"
  


#define WORKER_COUNT (1ULL << 16)
#define BLOCK_SIZE (256)
#define SEEDS_PER_CALL ((WORKER_COUNT) * (BLOCK_SIZE))
#define VERIFY false






#ifndef CHUNK_X
#define CHUNK_X 6
#endif
#ifndef CHUNK_Z
#define CHUNK_Z -3
#endif




#define MASK48 ((1ULL << 48) - 1ULL)
#define MASK32 ((1ULL << 32) - 1ULL)
#define MASK16 ((1ULL << 16) - 1ULL)

#define M1 25214903917ULL
#define ADDEND1 11ULL

#define M2 205749139540585ULL
#define ADDEND2 277363943098ULL

#define M4 55986898099985ULL
#define ADDEND4 49720483695876ULL

inline __host__ __device__ int64_t nextLong(uint64_t* seed) {
    *seed = (*seed * M1 + ADDEND1) & MASK48;
    int32_t u = *seed >> 16;
    *seed = (*seed * M1 + ADDEND1) & MASK48;
    return ((uint64_t)u << 32) + (int32_t)(*seed >> 16);
}

inline __device__ void addSeed(uint64_t seed, uint64_t* seeds, uint64_t* seedCounter)
{
    seeds[atomicAdd(seedCounter, 1)] = seed;
}

inline __host__ __device__ uint64_t makeMask(int32_t bits) {
    return (1ULL << bits) - 1;
}
// can use __builtin_ctz() on cpu and __device__​ int __clzll ( long long int x ) on gpu
inline __host__ __device__ int32_t countTrailingZeroes(uint64_t v) {
    int32_t c;

    v = (v ^ (v - 1)) >> 1;

    for(c = 0; v != 0; c++)  {
        v >>= 1;
    }

    return c;
}

inline __host__ __device__ uint64_t modInverse(uint64_t x) {
    uint64_t inv = 0;
    uint64_t b = 1;
    for (int32_t i = 0; i < 16; i++) {
        inv |= (1ULL << i) * (b & 1);
        b = (b - x * (b & 1)) >> 1;
    }
    return inv;
}



const uint64_t firstMultiplier = (M2 * CHUNK_X + M4 * CHUNK_Z) & MASK16;
__constant__ int32_t multTrailingZeroes;
__constant__ uint64_t firstMultInv;

__constant__ int32_t xCount;
__constant__ int32_t zCount;
__constant__ int32_t totalCount;


inline __host__ __device__ uint64_t getChunkSeed(uint64_t worldSeed) {
    uint64_t seed = (worldSeed ^ M1) & MASK48;
    int64_t a = nextLong(&seed) / 2 * 2 + 1;
    int64_t b = nextLong(&seed) / 2 * 2 + 1;
    return (uint64_t)(((CHUNK_X * a + CHUNK_Z * b) ^ worldSeed) & MASK48);
}

inline __host__ __device__ uint64_t getPartialAddend(uint64_t partialSeed, int32_t bits) {
    uint64_t mask = makeMask(bits);
    return ((uint64_t)CHUNK_X) * (((int32_t)(((M2 * ((partialSeed ^ M1) & mask) + ADDEND2) & MASK48) >> 16)) / 2 * 2 + 1) +
           ((uint64_t)CHUNK_Z) * (((int32_t)(((M4 * ((partialSeed ^ M1) & mask) + ADDEND4) & MASK48) >> 16)) / 2 * 2 + 1);
}

inline __device__ void addWorldSeed(uint64_t firstAddend, uint64_t c, uint64_t chunkSeed, uint64_t* seeds, uint64_t* seedCounter) {
    if(countTrailingZeroes(firstAddend) < multTrailingZeroes)
        return;
    uint64_t bottom32BitsChunkseed = chunkSeed & MASK32;

    uint64_t b = (((firstMultInv * firstAddend) >> multTrailingZeroes) ^ (M1 >> 16)) & makeMask(16 - multTrailingZeroes);
    if (multTrailingZeroes != 0) {
        uint64_t smallMask = makeMask(multTrailingZeroes);
        uint64_t smallMultInverse = smallMask & firstMultInv;
        uint64_t target = (((b ^ (bottom32BitsChunkseed >> 16)) & smallMask) -
                                (getPartialAddend((b << 16) + c, 32 - multTrailingZeroes) >> 16)) & smallMask;
        b += (((target * smallMultInverse) ^ (M1 >> (32 - multTrailingZeroes))) & smallMask) << (16 - multTrailingZeroes);
    }
    uint64_t bottom32BitsSeed = (b << 16) + c;
    uint64_t target2 = (bottom32BitsSeed ^ bottom32BitsChunkseed) >> 16;
    uint64_t secondAddend = (getPartialAddend(bottom32BitsSeed, 32) >> 16);
    secondAddend &= MASK16;
    uint64_t topBits = ((((firstMultInv * (target2 - secondAddend)) >> multTrailingZeroes) ^ (M1 >> 32)) & makeMask(16 - multTrailingZeroes));

    for (; topBits < (1ULL << 16); topBits += (1ULL << (16 - multTrailingZeroes))) {
        if (getChunkSeed((topBits << 32) + bottom32BitsSeed) == chunkSeed) {
            addSeed((topBits << 32) + bottom32BitsSeed, seeds, seedCounter);
        }
    }
}

__global__ void crack(uint64_t* in_buff, const uint64_t in_count, uint64_t* out_buff, uint64_t* out_count) {
    uint64_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id >= in_count)
        return;

    uint64_t chunkSeed = in_buff[global_id];
    int32_t x = CHUNK_X;
    int32_t z = CHUNK_Z;

	#if CHUNK_X == 0 && CHUNK_Z == 0
		addSeed(chunkSeed, out_buff, out_count);
	#else
		uint64_t f = chunkSeed & MASK16;
		uint64_t c = xCount == zCount ? chunkSeed & ((1ULL << (xCount + 1)) - 1) :
										chunkSeed & ((1ULL << (totalCount + 1)) - 1) ^ (1 << totalCount);
		#pragma unroll
		for (; c < (1ULL << 16); c += (1ULL << (totalCount + 1))) {
			uint64_t target = (c ^ f) & MASK16;
			uint64_t magic = (uint64_t)(x * ((M2 * ((c ^ M1) & MASK16) + ADDEND2) >> 16)) +
							 (uint64_t)(z * ((M4 * ((c ^ M1) & MASK16) + ADDEND4) >> 16));
			addWorldSeed(target - (magic & MASK16), c, chunkSeed, out_buff, out_count);
				#if CHUNK_X != 0
						addWorldSeed(target - ((magic + x) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
				#if CHUNK_Z != 0 && CHUNK_X != CHUNK_Z
						addWorldSeed(target - ((magic + z) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
				#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X + CHUNK_Z != 0
						addWorldSeed(target - ((magic + x + z) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
				#if CHUNK_X != 0 && CHUNK_X != CHUNK_Z
						addWorldSeed(target - ((magic + 2 * x) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
				#if CHUNK_Z != 0 && CHUNK_X != CHUNK_Z
						addWorldSeed(target - ((magic + 2 * z) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
				#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X + CHUNK_Z != 0 && CHUNK_X * 2 + CHUNK_Z != 0
						addWorldSeed(target - ((magic + 2 * x + z) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
				#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X != CHUNK_Z && CHUNK_X + CHUNK_Z != 0 && CHUNK_X + CHUNK_Z * 2 != 0
						addWorldSeed(target - ((magic + x + 2 * z) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
				#if CHUNK_X != 0 && CHUNK_Z != 0 && CHUNK_X + CHUNK_Z != 0
						addWorldSeed(target - ((magic + 2 * x + 2 * z) & MASK16), c, chunkSeed, out_buff, out_count);
				#endif
		}
	#endif // !(CHUNK_X == 0 && CHUNK_Z == 0)
}






#if defined(WIN32) || defined(_WIN32) || defined(__WIN32) && !defined(__CYGWIN__)
	#include <windows.h>
	uint64_t getCurrentTimeMillis() {
		SYSTEMTIME time;
		GetSystemTime(&time);
		return (uint64_t)((time.wSecond * 1000) + time.wMilliseconds);
	}
#else
	#include <sys/time.h>
	uint64_t getCurrentTimeMillis() {
		struct timeval te; 
		gettimeofday(&te, NULL); // get current time
		uint64_t milliseconds = te.tv_sec*1000LL + te.tv_usec/1000; // calculate milliseconds
		return milliseconds;
	}
#endif
#define GPU_ASSERT(code) gpuAssert((code), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
	fprintf(stderr, "GPUassert: %s (code %d) %s %d\n", hipGetErrorString(code), code, file, line);
	exit(code);
  }
}



uint64_t* inBuff;
uint64_t* outBuff;
uint64_t* outCount;

std::ifstream inFile;
std::ofstream outFile;

void setup() {
	hipSetDevice(0);
	GPU_ASSERT(hipPeekAtLastError());
	GPU_ASSERT(hipDeviceSynchronize());
	
	GPU_ASSERT(hipMallocManaged(&inBuff, sizeof(*inBuff) * SEEDS_PER_CALL));
	GPU_ASSERT(hipPeekAtLastError());
	
	GPU_ASSERT(hipMallocManaged(&outBuff, sizeof(*outBuff) * (SEEDS_PER_CALL)));
	GPU_ASSERT(hipPeekAtLastError());
	
	GPU_ASSERT(hipMallocManaged(&outCount, sizeof(*outCount)));
	GPU_ASSERT(hipPeekAtLastError());
	
	/*
	__constant__ int32_t multTrailingZeroes = countTrailingZeroes(firstMultiplier);
__constant__ uint64_t firstMultInv = modInverse(firstMultiplier >> multTrailingZeroes);

__constant__ int32_t xCount = countTrailingZeroes(CHUNK_X);
__constant__ int32_t zCount = countTrailingZeroes(CHUNK_Z);
__constant__ int32_t totalCount = countTrailingZeroes(CHUNK_X | CHUNK_Z);
*/
	auto tmp = countTrailingZeroes(firstMultiplier);
	GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(multTrailingZeroes), &tmp, sizeof(multTrailingZeroes)));
	GPU_ASSERT(hipPeekAtLastError());
	
	auto tmp2 = modInverse(firstMultiplier >> multTrailingZeroes);
	GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(firstMultInv), &tmp2, sizeof(firstMultInv)));
	GPU_ASSERT(hipPeekAtLastError());
	
	auto tmp3 = countTrailingZeroes(CHUNK_X);
	GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(xCount), &tmp3, sizeof(xCount)));
	GPU_ASSERT(hipPeekAtLastError());
	
	auto tmp4 = countTrailingZeroes(CHUNK_Z);
	GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(zCount), &tmp4, sizeof(zCount)));
	GPU_ASSERT(hipPeekAtLastError());
	
	auto tmp5 = countTrailingZeroes(CHUNK_X | CHUNK_Z);
	GPU_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(totalCount), &tmp5, sizeof(totalCount)));
	GPU_ASSERT(hipPeekAtLastError());
}

uint64_t fillBuffer() {
	uint64_t inCounter = 0;
	uint64_t curr_seed;
	for(inCounter = 0; inCounter < SEEDS_PER_CALL; inCounter++) {
		if (inFile >> curr_seed)
			inBuff[inCounter] = curr_seed;
		else
			break;
	}
	return inCounter;
}


int main() {
	inFile.open(INPUT_FILE);
	outFile.open(OUTPUT_FILE);
	setup();
	
	uint64_t in_buff_count = fillBuffer();
	while (in_buff_count != 0) {
		uint64_t start = millis();
		*outCount = 0;
		crack<<<WORKER_COUNT,BLOCK_SIZE>>>(inBuff, in_buff_count, outBuff, outCount);
		GPU_ASSERT(hipPeekAtLastError());
		GPU_ASSERT(hipDeviceSynchronize());
		
		
		if(VERIFY) {
			for (uint64_ {t outIndex = 0; outIndex < *outCount; outIndex++) {
				uint64_t chunkSeed = getChunkSeed(outBuff[outIndex]);
				bool match = false;
				for(uint64_t i = 0; i < in_buff_count; i++) {
					if (chunkSeed == inBuff[i]) {
						match = true;
						break;
					}
				}
				if (!match) {
					std::cout << "Seed: " << outBuff[outIndex] << " was not in original chunk seed list" << std::endl;
					exit(-1);
				}
			}
		}
		
		for (uint64_t outIndex = 0; outIndex < *outCount; outIndex++)
			outFile << outBuff[outIndex] << std::endl;
		
		std::cout << "Processed " << in_buff_count << " chunk seeds into " << *outCount << " world seeds in " << ((millis() - start)/1000) << " seconds" << std::endl;
		
		in_buff_count = fillBuffer();
	}
	outFile.close();
	std::cout << "Done" << std::endl;
	return 0;
}

